#include "hip/hip_runtime.h"
/*
 * 2019074 Jayraj Rathod
 * 2019203 Aniket Choudhari
 * 2019200 Aman Kumar
 */
#include <iostream>
#include <string>
#include <sstream>
#include <algorithm>
#include <stdlib.h>
#include <math.h>
#include <stdio.h>
#include <vector>
#include <thrust/scan.h>                                                        
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>

#include <fstream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include "cudaTriangleCounter.h"

#define BLOCK_SIZE 32

struct GlobalConstants {

    int *NodeList;
    int *ListLen;
    int numNodes;
    int numEdges;
};

__constant__ GlobalConstants cuConstCounterParams;

void
CudaTriangleCounter::setup() {

    int deviceCount = 0;
    std::string name;
    hipError_t err = hipGetDeviceCount(&deviceCount);

  

    // By this time the graph should be loaded.  Copying graph to 
    // data structures into device memory so that it is accessible to
    // CUDA kernels
    //

    hipMalloc(&cudaDeviceListLen, sizeof(int ) * numNodes);
    hipMemcpy(cudaDeviceListLen, list_len, sizeof(int) * numNodes, hipMemcpyHostToDevice);

    hipMalloc((void **)&cudaDeviceNodeList, node_list_size * sizeof(int));
    hipMemcpy(cudaDeviceNodeList, node_list, sizeof(int) * node_list_size, hipMemcpyHostToDevice);

    GlobalConstants params;
    params.ListLen = cudaDeviceListLen;
    params.NodeList = cudaDeviceNodeList;
    params.numNodes = numNodes;
    params.numEdges = numEdges;
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstCounterParams), &params, sizeof(GlobalConstants));
}

CudaTriangleCounter::CudaTriangleCounter(char *fileName) {
    clock_t start, diff, malloc_diff;
    int node, edge_id, temp = 0;
    int total_nodes = 0;
    int total_edges = 0;
    int msec;

    std::string line;
    std::ifstream myfile;
    myfile.open(fileName);

    std::string token;                                                             
    if (strstr(fileName,"new_orkut") != NULL) {                                    
        printf("This is the NEW_ORKUT FILE **\n");                             
        total_nodes = 3072600;                                                     
        total_edges = 117185083 + 1;                                               
    } else {                                                                       
        std::getline(myfile,line);                                                 
        std::stringstream lineStream(line);                                        
        while (lineStream >> token) {                                              
            if (temp == 0) {                                                       
                total_nodes = std::stoi(token, NULL, 10) + 1;                      
            } else if (temp == 1) {                                                
                total_edges = std::stoi(token, NULL, 10) + 1;                      
            } else {                                                               
                printf("!!!!!!!!!!!! TEMP IS %d\n ", temp);                        
                break;                                                             
            }                                                                      
            temp++;                                                                
        }                                                                          
    }

    start = clock();

    numNodes = total_nodes;
    node_list_size = total_edges * 2;
    numEdges = total_edges;

    printf("total_nodes %d\n", total_nodes);
    printf("node_list_size %d\n", node_list_size);
    printf("numEdges %d\n", numEdges);

    list_len = (int *)calloc(total_nodes, sizeof(int));
    start_addr = (int *)calloc(total_nodes, sizeof(int));
    node_list = (int *)calloc(node_list_size, sizeof(int));

    malloc_diff = clock() - start;
    msec = malloc_diff * 1000 / CLOCKS_PER_SEC;

    printf("memory allocated ......\n");
    node = 1;
    temp = 1;
    int neighbors;
    while(std::getline(myfile, line)) {
        neighbors = 0;
        std::stringstream lineStream(line);
        std::string token;
        while(lineStream >> token)
        {
            edge_id = std::stoi(token, NULL, 10);
            if (edge_id > node) {
                node_list[temp++] = edge_id;
                neighbors++;
            }
        }

        list_len[node] = neighbors;
        node++;
    }

    printf("graph created......\n");
    diff = clock() - start;
    msec = diff * 1000 / CLOCKS_PER_SEC;
    printf("time taken %d seconds %d milliseconds\n", msec/1000, msec%1000);

    myfile.close();
}

CudaTriangleCounter::~CudaTriangleCounter() {

    free(node_list);
    free(list_len);
}

void CudaTriangleCounter::countTriangles() {
    int i, j, k, m, count=0;

    for (i=1; i<numNodes; i++) {

        int *list = node_list + start_addr[i-1] + 1;

        int len = list_len[i];

        if (len < 2) {
            continue;
        }

        for (j=0; j<len-1; j++) {
            for (k=j+1; k<len; k++) {

                int idx1;
                int idx2;
                idx1 = list[j];
                idx2 = list[k];
                int *list1 = node_list + start_addr[idx1-1] + 1;
                int len1 = list_len[idx1];

                for (m=0; m<len1; m++) {

                    if (list1[m] == idx2) {
                        count++;
                    }
                }
            }

        }

    }
        printf("count for %d -> %d\n", i, count);

}

int main(int argc, char *argv[]) {

    if (argc != 2) {
        printf("usage: ./a.out <input_file>");
        exit(-1);
    }

    int msec;
    clock_t start, diff;

    CudaTriangleCounter *tCounter = new CudaTriangleCounter(argv[1]);
   
    tCounter->setup();
    start = clock();
    tCounter->countTriangles();
    diff = clock() - start;
    msec = diff * 1000 / CLOCKS_PER_SEC;
    // printf("counting taken %d seconds %d milliseconds\n", msec/1000, msec%1000);

    return 0;
}
